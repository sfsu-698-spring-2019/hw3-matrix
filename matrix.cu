
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <chrono>

// matrix multiply on gpu
__global__
void dgem_gpu(int n, float *A, float *B, float *C)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  // demo filler
  C[i+j*n] = B[i+j*n];
}

void square_dgemm_naive (int n, float* A, float* B, float* C)
{
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
        {
            float cij = C[i+j*n];
            for( int k = 0; k < n; k++ )
                cij += A[i+k*n] * B[k+j*n];
            C[i+j*n] = cij;
        }
}

int check(int n, float *A, float *B) {
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j) {
            double diff = std::abs(A[i + j * n] - B[i + j * n]);
            if (diff > 0.0003) {
                printf("diff is %f\n", diff);
                return 0;
            }
        }
    return 1;
}

int main(void)
{
  int N = 1000;
  int size = N*N; // square matrix
  float *A, *B, *C, *verify;

  // Works on cpu and gpu
  hipMallocManaged(&A, size*sizeof(float));
  hipMallocManaged(&B, size*sizeof(float));
  hipMallocManaged(&C, size*sizeof(float));
  hipMallocManaged(&verify, size*sizeof(float));


  // initialize x and y arrays on the host
  for (int i = 0; i < size; i++) {
    A[i] = 1.0f;
    B[i] = 2.0f;
    C[i] = 0.0f;
    verify[i] = 0.0f;
  }

  // this is to generate answer
  auto serialStart = std::chrono::system_clock::now();
  square_dgemm_naive(N, A, B, verify);
  auto serialEnd = std::chrono::system_clock::now();
  std::chrono::duration<double> serialElapsed = serialEnd - serialStart;
  std::cout << serialElapsed.count() << "s\n";

  // Run kernel on the GPU
  // use this one for actual work
  auto gpuStart = std::chrono::system_clock::now();
  // dgem_gpu<<<N, N>>>(N, A, B, C);
  // comment this one out, just for testing
  dgem_gpu<<<N, N>>>(N, A, C, verify);
  auto gpuEnd = std::chrono::system_clock::now();
  std::chrono::duration<double> gpuElapsed = gpuEnd - gpuStart;
  std::cout << gpuElapsed.count() << "s\n";

  // wait for threads to finish
  hipDeviceSynchronize();

  int correct = check(N, C, verify);

  // Free memory
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(verify);

  if (correct == 0) {
    printf("INVALID OUTPUT\n");
    exit(1);
  }

  printf("Correct output!\n");
  return 0;
}